#include<iostream>
#include<hip/hip_runtime.h>

__global__ void computeImageMean(const unsigned *image, const unsigned width, const unsigned height, float *result){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int imageIdx = row * width + col;
    atomicAdd(result, static_cast<float>(image[imageIdx]));
}

int main(){
    
    int width = 10;
    int height = 10;
    unsigned int *image = new unsigned int[height * width];
    for (int i = 0; i < height*width; i++){
        image[i] = i;
    }
    unsigned *image_cuda;
    hipMalloc((void **)&image_cuda, height*width*sizeof(unsigned int));
    hipMemcpy(image_cuda, image, height*width*sizeof(unsigned int), hipMemcpyHostToDevice);

    float *result_cuda;
    hipMalloc((void **)&result_cuda, sizeof(float));
    hipMemset(result_cuda, 0, sizeof(float));

    dim3 gridSize(2, 2);
    dim3 blockSize(width/gridSize.x, height/gridSize.y);
    computeImageMean<<<gridSize, blockSize>>>(image_cuda, width, height, result_cuda);


    float result = 0;
    hipMemcpy(&result, result_cuda, sizeof(float), hipMemcpyDeviceToHost);
    result = result/static_cast<float>(width*height);

    std::cout<<"result: "<<result<<std::endl;

    delete[] image;
    hipFree(image_cuda);
    hipFree(result_cuda);
}
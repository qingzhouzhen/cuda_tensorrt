
#include <iostream>
#include <hip/hip_runtime.h>

// 两个向量加法kernel，grid和block均为一维
__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
    // z[index] = x[index] + y[index];
}

int main()
{
    int N = 1 << 20;
    int nBytes = N * sizeof(float);
    // 申请host内存
    float *x, *y, *z;
    x = (float*)malloc(nBytes);
    y = (float*)malloc(nBytes);
    z = (float*)malloc(nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i)
    {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 申请device内存
    float *d_x, *d_y, *d_z;
    hipMalloc((void**)&d_x, nBytes);
    hipMalloc((void**)&d_y, nBytes);
    hipMalloc((void**)&d_z, nBytes);

    // 将host数据拷贝到device
    hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);
    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel

    hipEvent_t start, stop;
    hipEventCreate(&start);    
    hipEventCreate(&stop); 
    hipEventRecord(start); 

    add <<< gridSize, blockSize >>>(d_x, d_y, d_z, N);

    hipEventRecord(stop);               
    hipEventSynchronize(stop);             
    float time = 0;
    hipEventElapsedTime(&time, start, stop); 
    printf("Kernel execution time: %f milliseconds\n", time);
    hipEventDestroy(start);                
    hipEventDestroy(stop);

    // 将device得到的结果拷贝到host
    hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyDeviceToHost);

    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout<<z[0]<<std::endl;
    std::cout << "最大误差: " << maxError << std::endl;

    // 释放device内存
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    // 释放host内存
    free(x);
    free(y);
    free(z);

    return 0;
}

// 托管内存的写法
// int main()
// {
//     int N = 1 << 20;
//     int nBytes = N * sizeof(float);

//     // 申请托管内存
//     float *x, *y, *z;
//     cudaMallocManaged((void**)&x, nBytes);
//     cudaMallocManaged((void**)&y, nBytes);
//     cudaMallocManaged((void**)&z, nBytes);

//     // 初始化数据
//     for (int i = 0; i < N; ++i)
//     {
//         x[i] = 10.0;
//         y[i] = 20.0;
//     }

//     // 定义kernel的执行配置
//     dim3 blockSize(256);
//     dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
//     // 执行kernel
//     add << < gridSize, blockSize >> >(x, y, z, N);

//     // 同步device 保证结果能正确访问
//     cudaDeviceSynchronize();
//     // 检查执行结果
//     float maxError = 0.0;
//     for (int i = 0; i < N; i++)
//         maxError = fmax(maxError, fabs(z[i] - 30.0));
//     std::cout << "最大误差: " << maxError << std::endl;

//     // 释放内存
//     cudaFree(x);
//     cudaFree(y);
//     cudaFree(z);

//     return 0;
// }

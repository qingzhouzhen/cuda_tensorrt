
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel() {
    printf("Hello from thread %d, block %d\n", threadIdx.x, blockIdx.x);
}

int main() {
    // 定义执行配置和启动 kernel 函数
    myKernel<<<1, 5>>>();

    // 同步设备以确保 kernel 执行完成
    hipDeviceSynchronize();

    return 0;
}


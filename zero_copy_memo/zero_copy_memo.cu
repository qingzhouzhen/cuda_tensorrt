
#include <hip/hip_runtime.h>
#include <iostream>

// 对每个输入数据input每个元素加1并输出给result，grid和block均为一维
__global__ void add(float* input, float* result, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        result[i] = input[i] + 1;
    }
    // z[index] = x[index] + y[index];
}

int main(){


    hipSetDeviceFlags(hipDeviceMapHost);

    int N = 1 << 10;
    int size = N*sizeof(int); 
    // Allocate host memory using CUDA allocation calls
    float* h_in  = NULL;
    float* h_out = NULL;
    hipHostAlloc((void **)&h_in, size, hipHostMallocMapped);
    hipHostAlloc((void **)&h_out, size, hipHostMallocMapped);
    for(int i = 0; i < N; i++){
        h_in[i] = 1;
    }
    // Device arrays
    float *d_out, *d_in;

    // Get device pointer from host memory
    hipHostGetDevicePointer((void **)&d_in,  (void *) h_in , 0);
    hipHostGetDevicePointer((void **)&d_out, (void *) h_out, 0);


    // Launch the GPU kernel
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    add<<<gridSize, blockSize>>>(d_in, d_out, N);


    std::cout<<"output0: "<<h_out[0]<<std::endl;
    std::cout<<"output-1: "<<h_out[N-1]<<std::endl;

    // Continue processing on host using h_out
}


#include <hip/hip_runtime.h>
#include <iostream>

// 对每个输入数据input每个元素加1并输出给result，grid和block均为一维
__global__ void add(float* input, float* result, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        result[i] = input[i] + 1;
    }
    // z[index] = x[index] + y[index];
}

int main(){

    int N = 1 << 10;
    int size = N*sizeof(int); 
    // Host Arrays
    float* h_in  = new float[N];
    float* h_out = new float[N];
    for(int i = 0; i < N; i++){
        h_in[i] = 1;
    }
    std::cout<<"input0: "<<h_in[0]<<std::endl;
    std::cout<<"input-1: "<<h_in[N-1]<<std::endl;

    // Device arrays
    float *d_out, *d_in;

    // Allocate memory on the device
    hipMalloc((void **) &d_in,  size);
    hipMalloc((void **) &d_out, size);

    // Copy array contents of input from the host (CPU) to the device (GPU)
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    // Launch the GPU kernel
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    add<<<gridSize, blockSize>>>(d_in, d_out, N);

    // Copy result back
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    std::cout<<"output0: "<<h_out[0]<<std::endl;
    std::cout<<"output-1: "<<h_out[N-1]<<std::endl;

    // Continue processing on host using h_out
}

#include<iostream>
#include<hip/hip_runtime.h>
#define BLOCKSIZE 32
// 相同wrap中的线程访问不同bank的内存，无bank conflict
__global__ void kernel1(float *matrix, float *matrixTest, const int row, const int col){

    int x_id = blockDim.x * blockIdx.x + threadIdx.x; // 列坐标
    int y_id = blockDim.y * blockIdx.y + threadIdx.y; // 行坐标
    int index = y_id * col + x_id;

    __shared__ float sData[BLOCKSIZE][BLOCKSIZE+1];

    if (x_id < col && y_id < row){
        sData[threadIdx.x][threadIdx.y] = matrix[index];
        __syncthreads();
        matrixTest[index] = sData[threadIdx.x][threadIdx.y];
    }
}

int main(){

    int row = 32;
    int col = 32;
    float *matrixTest;
    hipMalloc((void **)&matrixTest, row*col*sizeof(float));
    hipMemset(matrixTest, 0, row*col*sizeof(float));
    float *matrix;
    hipMalloc((void **)&matrix, row*col*sizeof(float));
    hipMemset(matrix, 1, row*col*sizeof(float));


    dim3 gridSize(1, 1);
    dim3 blockSize(row, col);
    kernel1<<<gridSize, blockSize>>>(matrix, matrixTest, row, col);

    hipFree(matrixTest);
    hipFree(matrix);
}
